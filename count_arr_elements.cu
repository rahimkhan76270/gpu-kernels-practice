#include <hip/hip_runtime.h>

__global__ void count_equal_kernel(const int* input, int* output, int N, int K) {
    int idx=blockIdx.x*blockDim.x+threadIdx.x;
    if(idx<N)
    {
        if(input[idx]==K) atomicAdd(output,1);
    }
}

// input, output are device pointers (i.e. pointers to memory on the GPU)
extern "C" void solve(const int* input, int* output, int N, int K) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    hipMemset(output,0,sizeof(int));
    count_equal_kernel<<<blocksPerGrid, threadsPerBlock>>>(input, output, N, K);
    hipDeviceSynchronize();
}
