#include "hip/hip_runtime.h"
#include "solve.h"
#include <hip/hip_runtime.h>

__global__ void reverse_array(float* input, int N) {
    int i=blockIdx.x*blockDim.x+threadIdx.x;
    if(i<N/2)
    {
        float temp=input[i];
        input[i]=input[N-i-1];
        input[N-i-1]=temp;
    }
}

// input is device pointer
void solve(float* input, int N) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    reverse_array<<<blocksPerGrid, threadsPerBlock>>>(input, N);
    hipDeviceSynchronize();
}
